
#include <hip/hip_runtime.h>
__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void wrap(float *x, float *y, int N) {

    float *d_x, *d_y;

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));


  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);


  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

}
